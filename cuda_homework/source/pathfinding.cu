#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_cooperative_groups.h>

#include "errors.h"
#include "pathfinding.cuh"
#include "hashtable.cuh"
#include "heap.cuh"

using namespace cooperative_groups;

__constant__ Coord endCuda;

Pathfinding::Pathfinding (const Config& config) : config(config) {
  FILE* input = fopen(config.input_data.c_str(), "r");

  fscanf(input, "%d,%d", &n, &m);
  fscanf(input, "%d,%d", &start.x, &start.y);
  fscanf(input, "%d,%d", &end.x, &end.y);

  gridHost = (int*) malloc(sizeof(int) * n * m);
  if (gridHost == nullptr) {
    fprintf(stderr, "Memory allocation failed!\n");
    exit(1);
  }

  for (int i = 0; i < n * m; i++) {
    gridHost[i] = 1;
  }

  int holes, non_ones;

  fscanf(input, "%d", &holes); 
  for (int i = 0; i < holes; i++) {
    int x, y;
    fscanf(input, "%d,%d", &x, &y);
    gridHost[getPosition(x, y)] = -1;
  }

  fscanf(input, "%d", &non_ones);
  for (int i = 0; i < non_ones; i++) {
    int x, y, val;
    fscanf(input, "%d,%d,%d", &x, &y, &val);
    gridHost[getPosition(x, y)] = val;
  }
}

Pathfinding::~Pathfinding() {
  if (gridHost != nullptr) {
    free(gridHost);
  }

  if (gridCuda != nullptr) {
    hipFree(gridCuda);
  }

  if (statesHost != nullptr) {
    free(statesHost);
  }

  if (statesCuda != nullptr) {
    hipFree(statesCuda);
  }

  if (queuesCuda != nullptr) {
    hipFree(queuesCuda);
  }

  if (queueSizesCuda != nullptr) {
    hipFree(queueSizesCuda);
  }

  if (hashtableCuda != nullptr) {
    hipFree(hashtableCuda);
  }
}

__device__ void Pathfinding::lock() {
  if (threadIdx.x == 0) {
    lockCuda.lock();
  }
  __syncthreads();
}

__device__ void Pathfinding::unlock() {
  if (threadIdx.x == 0) {
    lockCuda.unlock();
  }
  __syncthreads();
}

__device__ bool Pathfinding::inBounds(int x, int y) {
  return x >= 0 && x < n && y >= 0 && y < m;
}

__device__ void Pathfinding::expand(State& st, int stateIdx, int firstFreeSlot) {
  if (st.isNull()) {
    return;
  }

  int x = st.node % n;
  int y = st.node / n;
  
  int idx = firstFreeSlot;

  for (int i : {-1, 0, 1}) {
    for (int j : {-1, 0, 1}) {
      if (i == 0 && j == 0) continue;

      int nx = x + i;
      int ny = y + j;

      if (inBounds(nx, ny)) {
        int newNode = getPosition(nx, ny);
        statesCuda[idx].prev = stateIdx;
        statesCuda[idx].node = newNode;
        if (gridHost[newNode] != -1) {
          statesCuda[idx].g = st.g + gridHost[newNode];
          statesCuda[idx].f = statesCuda[idx].g + abs(nx - endCuda.x) 
                            + abs(ny - endCuda.y);
        }
      }

      idx++;
    }
  }
}

__device__ void Pathfinding::extract() {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int blockOffset = blockIdx.x * blockDim.x;

  __shared__ int offsets[THREADS_PER_BLOCK];

  lock();

  if (threadIdx.x == 0) {
    offsets[0] = *statesSizeCuda;
    for (int i = 1; i < THREADS_PER_BLOCK; i++) {
      offsets[i] = offsets[i-1] + 8 * max(8, queueSizesCuda[i - 1 + blockOffset]);
    }
    *statesSizeCuda = offsets[THREADS_PER_BLOCK - 1] +
                      8 * max(8, queueSizesCuda[THREADS_PER_BLOCK - 1 +
                          blockOffset]);
  }

  unlock();

  int firstFreeSlot = offsets[threadIdx.x];

  for (int i = 0; i < 8 && !empty(queueSizesCuda[idx]); i++) {
    QState qst = pop(queuesCuda + HEAP_SIZE * idx, queueSizesCuda[idx]);
    State st = statesCuda[qst.stateNumber];
    expand(st, qst.stateNumber, firstFreeSlot);
    firstFreeSlot += 8;
  }
}

__device__ void Pathfinding::findPath() {
  grid_group grid = this_grid();

  while(true || !*finishedCuda) {
    extract();
    if (threadIdx.x == 0 && blockDim.x == 0) {
      int finished = true;
      for (int i = 0; i < BLOCKS * THREADS_PER_BLOCK; i++) {
        if (queueSizesCuda[i] > 0) {
          finished = false;
          break;
        }
      }
      if (finished) {
        *finishedCuda = true;
      }
    }
    grid.sync();
  }
}

__global__ void kernel(Pathfinding* pathfinding) {
  pathfinding->findPath();
}


void Pathfinding::solve() {
  printf("n: %d, m: %d\n", n, m);
  printf("start: %d, %d\n", start.x, start.y);
  printf("end: %d, %d\n", end.x, end.y);
  printGrid();

  HANDLE_ERROR(hipMalloc(&gridCuda, sizeof(State) * n * m));
  HANDLE_ERROR(hipMalloc(&statesCuda, sizeof(State) * n * m));
  HANDLE_ERROR(hipMalloc(&queuesCuda, sizeof(State) * BLOCKS * HEAP_SIZE));
  HANDLE_ERROR(hipMalloc(&queueSizesCuda, sizeof(int) * BLOCKS *
        QUEUES_PER_BLOCK));
  HANDLE_ERROR(hipMalloc(&hashtableCuda, sizeof(int) * TABLE_SIZE));
  HANDLE_ERROR(hipMalloc(&statesSizeCuda, sizeof(int)));
  HANDLE_ERROR(hipMalloc(&finishedCuda, sizeof(bool)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(endCuda), &end, sizeof(Coord)));

  // TODO: handle errors
  statesHost = (State*) malloc(sizeof(State) * n * m);

  for (int i = 0; i < n * m; i++) {
    statesHost[i] = State();
  }

  int startNode = getPosition(start.x, start.y);
  State initState = {
    .f = abs(start.x - end.x) + abs(start.y - end.y),
    .g = 0,
    .prev = -1,
    .node = startNode,
  };
  QState initQState = {
    .f = 0,
    .stateNumber = 0,
  };

  statesHost[startNode] = initState;

  HANDLE_ERROR(hipMemcpy(statesCuda, statesHost, sizeof(State) * n * m,
        hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(queuesCuda, &initQState, sizeof(QState),
        hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset(queueSizesCuda, 0, sizeof(int) * BLOCKS *
        QUEUES_PER_BLOCK));
  HANDLE_ERROR(hipMemset(statesSizeCuda, 0, sizeof(int)));
  HANDLE_ERROR(hipMemset(finishedCuda, 0, sizeof(bool)));

  int one = 1;
  HANDLE_ERROR(hipMemcpy(queueSizesCuda, &one, sizeof(int),
        hipMemcpyHostToDevice));

  // TODO: add timing


  void *kernelArgs[] = {(void*) this};
  HANDLE_ERROR(hipLaunchCooperativeKernel((void*) kernel, BLOCKS,
        THREADS_PER_BLOCK, kernelArgs));

  printf("Computation finished!\n");
                              

  // TODO: copy back the results
  // TODO: recreate the path
}


