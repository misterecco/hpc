#include "hip/hip_runtime.h"
#include <assert.h>

#include "pathfinding.cuh"
#include "hashtable.cuh"
#include "heap.cuh"

Pathfinding::Pathfinding (const Config& config) : config(config) {
  FILE* input = fopen(config.input_data.c_str(), "r");

  fscanf(input, "%d,%d", &n, &m);
  fscanf(input, "%d,%d", &start.x, &start.y);
  fscanf(input, "%d,%d", &end.x, &end.y);

  gridHost = (int*) malloc(sizeof(int) * n * m);
  if (gridHost == nullptr) {
    fprintf(stderr, "Memory allocation failed!\n");
    exit(1);
  }

  for (int i = 0; i < n * m; i++) {
    gridHost[i] = 1;
  }

  int holes, non_ones;

  fscanf(input, "%d", &holes); 
  for (int i = 0; i < holes; i++) {
    int x, y;
    fscanf(input, "%d,%d", &x, &y);
    gridHost[getPosition(x, y)] = -1;
  }

  fscanf(input, "%d", &non_ones);
  for (int i = 0; i < non_ones; i++) {
    int x, y, val;
    fscanf(input, "%d,%d,%d", &x, &y, &val);
    gridHost[getPosition(x, y)] = val;
  }
}

Pathfinding::~Pathfinding() {
  if (gridHost != nullptr) {
    free(gridHost);
  }

  if (gridCuda != nullptr) {
    hipFree(gridCuda);
  }

  if (statesHost != nullptr) {
    free(statesHost);
  }

  if (statesCuda != nullptr) {
    hipFree(statesCuda);
  }

  if (queuesCuda != nullptr) {
    hipFree(queuesCuda);
  }

  if (queueSizesCuda != nullptr) {
    hipFree(queueSizesCuda);
  }

  if (hashtableCuda != nullptr) {
    hipFree(hashtableCuda);
  }
}

__global__ void kernel() {
  return;
}

void Pathfinding::solve() {
  printf("n: %d, m: %d\n", n, m);
  printf("start: %d, %d\n", start.x, start.y);
  printf("end: %d, %d\n", end.x, end.y);
  printGrid();

  // TODO: handle errors
  hipMalloc(&gridCuda, sizeof(State) * n * m);
  hipMalloc(&statesCuda, sizeof(State) * n * m);
  hipMalloc(&queuesCuda, sizeof(State) * BLOCKS * HEAP_SIZE);
  hipMalloc(&queueSizesCuda, sizeof(int) * BLOCKS * QUEUES_PER_BLOCK);
  hipMalloc(&hashtableCuda, sizeof(int) * TABLE_SIZE);

  // TODO: handle errors
  statesHost = (State*) malloc(sizeof(State) * n * m);

  for (int i = 0; i < n * m; i++) {
    statesHost[i] = State();
  }

  int startNode = getPosition(start.x, start.y);
  State initState = {
    .f = 0,
    .g = 0,
    .prev = -1,
    .node = startNode,
  };
  QState initQState = {
    .f = 0,
    .stateNumber = 0,
  };

  statesHost[startNode] = initState;

  // TODO: handle errors
  hipMemcpy(statesCuda, statesHost, sizeof(State) * n * m, hipMemcpyHostToDevice);
  hipMemcpy(queuesCuda, &initQState, sizeof(QState), hipMemcpyHostToDevice);
  hipMemset(queueSizesCuda, 0, sizeof(int) * BLOCKS * QUEUES_PER_BLOCK);

  int one = 1;
  hipMemcpy(queueSizesCuda, &one, sizeof(int), hipMemcpyHostToDevice);
}

