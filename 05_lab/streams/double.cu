#include "hip/hip_runtime.h"
#include "./common/helpers.h"

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    if (tid < N) {
        int tid1 = (tid + 1) % 256;
        int tid2 = (tid + 2) % 256;
        float aSum = (a[tid] + a[tid1] + a[tid2]) / 3.0f;
        float bSum = (b[tid] + b[tid1] + b[tid2]) / 3.0f;
        c[tid] = (aSum + bSum) / 2;
    }
}

int main(void) {
    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t streams[2];
    for (int i = 0; i < 2; i++) {
        hipStreamCreate(&(streams[i]));
    }

    int *host_a, *host_b, *host_c;
    int *dev_a[2], *dev_b[2], *dev_c[2];

    for (int i = 0; i < 2; i++) {
	HANDLE_ERROR(hipMalloc((void**)&(dev_a[i]), N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&(dev_b[i]), N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&(dev_c[i]), N * sizeof(int)));
    }

    HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    for (int i = 0, j = 0; i < FULL_DATA_SIZE; i += N, j++) {
        const hipStream_t& stream = streams[j%2];
        HANDLE_ERROR(hipMemcpyAsync(dev_a[j%2], host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_b[j%2], host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

        kernel<<<N / 256, 256, 0, stream>>>(dev_a[j%2], dev_b[j%2], dev_c[j%2]);

        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c[j%2], N * sizeof(int), hipMemcpyDeviceToHost, stream));
    }

    for (int i = 0; i < 2; i++) {
    	HANDLE_ERROR(hipStreamSynchronize(streams[i]));
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
   

    for (int i = 0; i < 2; i++) {
        HANDLE_ERROR(hipFree(dev_a[i]));
        HANDLE_ERROR(hipFree(dev_b[i]));
        HANDLE_ERROR(hipFree(dev_c[i]));
    	HANDLE_ERROR(hipStreamDestroy(streams[i]));
    }
    
    return 0;
}
